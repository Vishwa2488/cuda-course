#include <hip/hip_runtime.h>
#include <stdio.h>

#include <iostream>

#define CUDA_ERROR_CHECK(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char * const filename, const int line)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "[ERROR] %d, %s %s - %s\n", line, filename, func, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


__global__
void kernel1(float * A, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n)
    {
        A[idx] *= 2;
    }
    return;
}

__global__
void kernel2(float * A, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n)
    {
        A[idx] += 1;
    }

    return;
}



void CUDART_CB myStreamCallback(hipStream_t stream1,hipError_t status, void *userData)
{
    printf("Stream completed successfully\n");
    return;
}

int main()
{
    float * d_data, * h_data;
    int N = 256;
    int size = N * sizeof(float);

    hipStream_t stream1, stream2;
    hipEvent_t event;

    CUDA_ERROR_CHECK(hipStreamCreate(&stream1));
    CUDA_ERROR_CHECK(hipStreamCreate(&stream2));
    CUDA_ERROR_CHECK(hipEventCreate(&event));
    CUDA_ERROR_CHECK(hipMalloc(&d_data, size));
    CUDA_ERROR_CHECK(hipHostMalloc(&h_data, size, hipHostMallocDefault));
    for (int i=0;i<N;i++)
    {
        h_data[i] = static_cast <float> (i);
    }

    CUDA_ERROR_CHECK(hipMemcpyAsync(d_data, h_data, size, hipMemcpyHostToDevice, stream1));
    kernel1 <<<(N + 255)/256, 256, 0, stream1>>> (d_data, N);

    CUDA_ERROR_CHECK(hipStreamSynchronize(stream1)); // not reqd as the event recording for stream1 ensures this

    CUDA_ERROR_CHECK(hipEventRecord(event, stream1)); // waits till all threads of stream1 completes

    CUDA_ERROR_CHECK(hipStreamWaitEvent(stream2, event, 0)); // waits for the event

    kernel2 <<<(N+255)/256, 256, 0, stream2>>> (d_data, N);

    CUDA_ERROR_CHECK(hipMemcpyAsync(h_data, d_data, size, hipMemcpyDeviceToHost, stream2));
    
    CUDA_ERROR_CHECK(hipStreamSynchronize(stream1));
    CUDA_ERROR_CHECK(hipStreamSynchronize(stream2));

    CUDA_ERROR_CHECK(hipStreamAddCallback(stream2, myStreamCallback, NULL, 0));
    bool correct = true;

    for (int i=0;i<N;i++)
    {
        if (fabs(h_data[i] - (static_cast <float> (i) * 2 + 1)) > 1e-5)
        {
            correct = false;
            break;
        }
    }

    printf("Answer %s\n", (correct)? "correct": "incorrect");

    CUDA_ERROR_CHECK(hipFree(d_data));
    CUDA_ERROR_CHECK(hipHostFree(h_data));
    CUDA_ERROR_CHECK(hipStreamDestroy(stream1));
    CUDA_ERROR_CHECK(hipStreamDestroy(stream2));
    CUDA_ERROR_CHECK(hipEventDestroy(event));

    return 0;
}